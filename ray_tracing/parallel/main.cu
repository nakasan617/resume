#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#undef __STRICT_ANSI__
#include <math.h>
//#include <time.h> 
//#include <omp.h>
//#include "my_vector.h"
#include "atomic_add.h"

#define W_MAX 10

double ** allocate_G(int n) {
	double ** rv = (double **)calloc(n, sizeof(double *));
	for(int i = 0; i < n; i++) {
		rv[i] = (double *)calloc(n, sizeof(double));
	}
	
	return rv;
}

__device__ double LCG_random_double(uint64_t * seed) {
	const uint64_t m = 9223372036854775808ULL;
	const uint64_t a = 2806196910506780709ULL;
	const uint64_t c = 1ULL;
	*seed = (a* (*seed) + c) % m;
	return (double) (*seed)/ (double)m;
}

__device__ int translation(double a, double start, double end, double N, int axis) {
	double cell_size = (end - start)/N;
	if( cell_size < 0 ) 
		cell_size *= -1;
	int rv = 0;
	assert(axis == 0 || axis == 1);
	if( axis == 0 ) {
		while(!(start <= a && a < start + cell_size)) { 
			rv++;
			start += cell_size;
			assert(start <= end);
		}
	} else {
		while(!(start >= a && a > start - cell_size)) {
			rv++;
			start -= cell_size;
			assert(start >= end);
		} 
	}
	return rv;
}

__device__ double inner(double A [], double B [], int length);
__device__ static inline int condition_fast(double W_x, double W_z, double V [], double C [], double R) {
	//printf("this is W_x %lf, W_z %lf \n", W_x, W_z);
	if( (-1 * W_MAX < W_x) && (W_x < W_MAX) && (-1 * W_MAX < W_z) && (W_z < W_MAX)) {
		double inner_product = inner(V, C, 3);
		double norm = inner(C, C, 3);
		//printf("this is condition %lf\n", inner * inner + R * R - norm);
		if( inner_product * inner_product + R * R - norm > 0) {
			return 1; 
		}
		return 0;
	}
	return 0;
}


__device__ double inner(double A [], double B [], int length) {
	double rv = 0;
	for(int i = 0; i < length; i++) {
		rv += A[i] * B[i];
	}
	return rv;
}

__global__ void one_iteration_fast(int n, double * G_D) {
	double W_y = 10;
	double R = 6;

	double L [3]; 
	double C [3];
	L[0] = 4; L[1] = 4; L[2] = -1;
	C[0] = 0; C[1] = 12; C[2] = 0;

	double phi = -1;
	double cos_theta = -2;
	double sin_theta = -2;

	double V [3];
	double W [3];
	double I [3];
	double N [3];
	double S [3];

	double W_x = -1;
	double W_z = -1;
	double t = 0;
	double b = -1;
	int i = 0;
	int j = 0;
	uint64_t seed = (blockDim.x * blockIdx.x + threadIdx.x) * 4238811;
	//assert(0 == 1);

	do { 
		phi = 2 * M_PI * LCG_random_double(&seed); // this should be 0 to 2 pi 			
		cos_theta = 2 * LCG_random_double(&seed) - 1; // this should be -1 to 1
		sin_theta = sqrt( 1 - cos_theta * cos_theta);
		assert(0 < phi && phi < 2 * M_PI); assert(-1 < cos_theta && cos_theta < 1); assert(-1 <= sin_theta && sin_theta <= 1);
		V[0] = sin_theta * cos(phi); V[1] = sin_theta * sin(phi); V[2] = cos_theta; 
		W[0] = V[0] * W_y/V[1]; W[1] = V[1] * W_y/V[1]; W[2] = V[2] * W_y/V[1];
		W_x = W[0];
		W[1] = W_y;
		W_z = W[2];
	} while (condition_fast(W_x, W_z, V, C, R) == 0);
	double a = inner(V, C, 3);
	assert(inner(C, C, 3) >= 0);
	t = a - sqrt(a * a + R * R - inner(C, C, 3));
	I[0] = t * V[0]; I[1] = t * V[1]; I[2] = t * V[2];

	// calculating N here 
	N[0] = I[0] - C[0]; N[1] = I[1] - C[1]; N[2] = I[2] - C[2];
	a = sqrt(inner(N, N, 3));
	assert(a != 0);
	N[0] = 1/a * N[0]; N[1] = 1/a * N[1]; N[2] = 1/a * N[2];

	// calculating S here
	S[0] = L[0] - I[0]; S[1] = L[1] - I[1]; S[2] = L[2] - I[2];
	a = sqrt(inner(S, S, 3));
	assert(a != 0);
	S[0] = 1/a * S[0]; S[1] = 1/a * S[1]; S[2] = 1/a * S[2];

	// change b accordingly
	b = inner(S, N, 3);
	if(b < 0) b = 0;

	i = translation(W_x, W_MAX, -1 * W_MAX, n, 1);
	j = translation(W_z, -1 * W_MAX, W_MAX, n, 0);
	assert(0 <= i); assert(i < n); assert(0 <= j); assert(j < n);
	atomicAdd_(&(G_D[i * n + j]), b);
}

int main(int argc, char ** argv) {
	assert(argc == 4);
	int n = atoi(argv[1]);
	int num_block = atoi(argv[2]);
	int threads_per_block = atoi(argv[3]);
	

	double * G = (double *)malloc(n*n *sizeof(double));
	for(int i = 0; i < n*n; i++) {
		G[i] = 0;
	}
	//G = allocate_G(n);
	double * G_D;
	assert(hipMalloc((void **)&G_D, sizeof(double) * n * n) == hipSuccess);
	//assert(G[n * n - 1] == 0);

	for(int i = 0; i < n * n; i++ ) {
		assert(hipMemcpy(&(G_D[i]), &(G[i]), sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
	}

	//printf("going through the iterations with %d blocks with %d threads\n", num_block, threads_per_block);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	one_iteration_fast<<<num_block, threads_per_block>>>(n, G_D);
	hipError_t err;
	if((err = hipGetLastError()) != hipSuccess) { 
		const char * str = hipGetErrorName(err);
		printf("%s\n", str);
	}
	hipEventRecord(stop);

	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			err = hipMemcpy(G + i * n + j,G_D + i * n + j, sizeof(double),hipMemcpyDeviceToHost);
			if(err != hipSuccess){
				const char * str = hipGetErrorName(err);
				printf("%s\n", str);
			}
		}
	}
	
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("it took %f microseconds when %d blocks and %d threads are used\n", milliseconds, num_block, threads_per_block);

	// print to out.csv
	
	FILE * fp = fopen("out.csv", "w");
	//FILE * fp = stdout;
	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			if( j == n -1) {
				fprintf(fp, "%lf\n", G[i*n + j]);
			} else {
				fprintf(fp, "%lf, ", G[i*n + j]);
			}
		}
	}
		
	fclose(fp);
	free(G);
	hipFree(G_D);
	exit(0);
}
